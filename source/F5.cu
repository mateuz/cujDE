#include "hip/hip_runtime.h"
#include "F5.cuh"
#include "IO.h"
#include "constants.cuh"

#include <iostream>
#include <vector>
#include <iterator>
#include <fstream>

/*
 * Shifted and Rotated Rosenbrock's Function
 *
 * as defined in "Problem Definitions and Evaluation Criteria for the
 * CEC 2013 Special Session and Competition on Real-Parameter Optimization",
 * by Liang, J.J., Qu, B.-Y., Suganthan, P.N., Hernandez-Diaz, A.G.,
 * Computational Intelligence Laboratory, Zhengzhou University, Zhengzhou,
 * China and Nanyang Technological University, Singapore, Technical Report,
 * v. 2012, p. 3-18, 2013.
*/

F5::F5(uint _dim, uint _ps):Benchmarks()
{
  n_dim = _dim;
  ps = _ps;
  min = -100.0;
  max = +100.0;
  ID = 5;

  n_threads = 32;
  n_blocks = (ps%n_threads)? (ps/n_threads)+1 : (ps/n_threads);

  /* ---------------------------------------------- */
  /* Load a shift vector to test the bench function */
  std::string file_name = "data-files/shift_rosenbrock.mat";
  std::string vec_name = "Shift - Rosenbrock [-100.0, +100.0]";
  IO * io = new IO();
  std::ifstream file(file_name);
  if( not file.is_open() ){
    std::cout << "\"data-files/shift_rosenbrock.mat\" could not be opened\n";
    exit(-1);
  }
  auto loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(shift), (void *) loaded_vec.data(), n_dim * sizeof(float)));

  /* ---------------------------------------------- */
  /* Load a rotate matrix                           */
  if(!(n_dim==2 or n_dim==5 or n_dim==10 or n_dim==20 or n_dim==30 or n_dim==50 or n_dim==100)){
    printf("\nError: Rotation matrix are only defined for D = 2,5,10,20,30,50,100.\n");
    exit(-1);
  }

  file_name = "data-files/rot/M_D" + std::to_string(n_dim) + ".txt";
  vec_name = "M_D" + std::to_string(n_dim);
  file.open(file_name, std::ifstream::in);
  if( not file.is_open() ){
    std::cout << "Error opening rotation matrix file\n";
    exit(-1);
  }
  loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_rotation), (void *) loaded_vec.data(), n_dim * n_dim * sizeof(float)));

}

F5::~F5()
{
  /*empty*/
}

__global__ void computeK_F5(float * x, float * f){
  uint id_p = threadIdx.x + (blockIdx.x * blockDim.x);
  uint ps = params.ps;
  if( id_p < ps ){
    uint ndim = params.n_dim;
    uint id_d = id_p * ndim;
    uint i, j;

    //The constant 2.048/100 is needed because on rotate operation
    //the value of a dimension can be higher than bounds;

    float z[100];
    //shift
    for( i = 0; i < ndim; i++ )
      z[i] = (x[id_d + i] - shift[i]) * 2.048/100;

    float z_rot[100];
    //rotation
    for( i = 0; i < ndim; i++ ){
      z_rot[i] = 0.0;
      for( j = 0; j < ndim; j++ )
        z_rot[i] += z[j] * m_rotation[i * ndim + j];
      z_rot[i] += 1.0;
    }

    float s = 0.0, t1, t2;
    for(uint i = 0; i < (ndim - 1); i++){
      t1 = z_rot[i+1] - (z_rot[i] * z_rot[i]);
      t2 = z_rot[i] - 1.0;

      t1 *= t1;
      t2 *= t2;

      s += (100.0 * t1) + t2;
    }

    if( s <= 10e-08 )
      s = 0.0;
    f[id_p] = s;
  }
}

void F5::compute(float * x, float * f){
  computeK_F5<<< n_blocks, n_threads >>>(x, f);
  checkCudaErrors(hipGetLastError());
}
