#include "hip/hip_runtime.h"
#include "F5.cuh"
#include "IO.h"
#include "constants.cuh"

#include <iostream>
#include <vector>
#include <iterator>
#include <fstream>

/*
 * Shifted and Rotated Rosenbrock's Function
 *
 * as defined in "Problem Definitions and Evaluation Criteria for the
 * CEC 2013 Special Session and Competition on Real-Parameter Optimization",
 * by Liang, J.J., Qu, B.-Y., Suganthan, P.N., Hernandez-Diaz, A.G.,
 * Computational Intelligence Laboratory, Zhengzhou University, Zhengzhou,
 * China and Nanyang Technological University, Singapore, Technical Report,
 * v. 2012, p. 3-18, 2013.
*/

F5::F5(uint _dim, uint _ps):Benchmarks()
{
  n_dim = _dim;
  ps = _ps;
  min = -100.0;
  max = +100.0;
  ID = 5;

  n_threads = 32;
  n_blocks = (ps%n_threads)? (ps/n_threads)+1 : (ps/n_threads);

  /* ---------------------------------------------- */
  /* Load a shift vector to test the bench function */
  std::string file_name = "data-files/shift_rosenbrock.mat";
  std::string vec_name = "Shift - Rosenbrock [-100.0, +100.0]";
  IO * io = new IO();
  std::ifstream file(file_name);
  if( not file.is_open() ){
    std::cout << "\"data-files/shift_rosenbrock.mat\" could not be opened\n";
    exit(-1);
  }
  auto loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(shift), (void *) loaded_vec.data(), n_dim * sizeof(float)));

  /* ---------------------------------------------- */
  /* Load a rotate matrix                           */
  if(!(n_dim==2 or n_dim==5 or n_dim==10 or n_dim==20 or n_dim==30 or n_dim==50 or n_dim==100)){
    printf("\nError: Rotation matrix are only defined for D = 2,5,10,20,30,50,100.\n");
    exit(-1);
  }

  file_name = "data-files/rot/M_D" + std::to_string(n_dim) + ".txt";
  vec_name = "M_D" + std::to_string(n_dim);
  file.open(file_name, std::ifstream::in);
  if( not file.is_open() ){
    std::cout << "Error opening rotation matrix file\n";
    exit(-1);
  }
  loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_rotation), (void *) loaded_vec.data(), n_dim * n_dim * sizeof(float)));

}

F5::~F5()
{
  /*empty*/
}


__global__ void computeK2_F5(float * x, float * f){
  uint id_p, id_d, ps, ndim, i, stride;

  id_p = blockIdx.x;
  id_d = threadIdx.x;
  ps = params.ps;
  ndim = params.n_dim;
  stride = id_p * ndim;

  float a, b, t1, t2;

  __shared__ float r[128];
  __shared__ float z[100];
  __shared__ float R[10000];
  __shared__ float z_rot[100];

  r[id_d] = 0.0f;

  if( id_d < ndim ){
    z[id_d] = (x[stride+id_d] - shift[id_d]) * 0.02048;
    //each dimension load your rotation column from rotation matrix
    for( i = 0; i < ndim; i++ ){
      R[(id_d*ndim)+i] = m_rotation[(id_d*ndim)+i];
    }
  }

  __syncthreads();

  if( id_d < ndim ){
    z_rot[id_d] = 0.0;
    for( i = 0; i < ndim; i++ ){
      z_rot[id_d] += z[i] * R[(id_d*ndim)+i];
    }
    z_rot[id_d] += 1.0;
  }

  __syncthreads();

  if( id_d < (ndim-1) ){
    a = z_rot[id_d];
    b = z_rot[id_d+1];
    t1 = b - (a * a);
    t2 = a - 1.0;

    t1 *= t1;
    t2 *= t2;

    r[id_d] = (100.0 * t1) + t2;

    __syncthreads();

    /* Simple reduce sum */
    if( id_d < 64 )
      r[id_d] += r[id_d + 64];

    __syncthreads();

    if( id_d < 32 )
      r[id_d] += r[id_d + 32];

    __syncthreads();

    if( id_d < 16 )
      r[id_d] += r[id_d + 16];

    __syncthreads();

    if( id_d < 8 )
      r[id_d] += r[id_d + 8];

    __syncthreads();

    if( id_d < 4 )
      r[id_d] += r[id_d + 4];

    __syncthreads();

    if( id_d < 2 )
      r[id_d] += r[id_d + 2];

    __syncthreads();

    if( id_d == 0 )
      r[id_d] += r[id_d + 1];

    __syncthreads();

    if( id_d == 0 )
      f[id_p] = r[0];
  }
}

__global__ void computeK_F5(float * x, float * f){
  uint id_p = threadIdx.x + (blockIdx.x * blockDim.x);
  uint ps = params.ps;
  if( id_p < ps ){
    uint ndim = params.n_dim;
    uint id_d = id_p * ndim;
    uint i, j;

    //The constant 2.048/100 is needed because on rotate operation
    //the value of a dimension can be higher than bounds;

    float z[100];
    //shift
    for( i = 0; i < ndim; i++ )
      z[i] = (x[id_d + i] - shift[i]) * 2.048/100;

    float z_rot[100];
    //rotation
    for( i = 0; i < ndim; i++ ){
      z_rot[i] = 0.0;
      for( j = 0; j < ndim; j++ )
        z_rot[i] += z[j] * m_rotation[i * ndim + j];
      z_rot[i] += 1.0;
    }

    float s = 0.0, t1, t2;
    for(uint i = 0; i < (ndim - 1); i++){
      t1 = z_rot[i+1] - (z_rot[i] * z_rot[i]);
      t2 = z_rot[i] - 1.0;

      t1 *= t1;
      t2 *= t2;

      s += (100.0 * t1) + t2;
    }

    if( s <= 10e-08 )
      s = 0.0;
    f[id_p] = s;
  }
}

void F5::compute(float * x, float * f){
  computeK_F5<<< n_blocks, n_threads >>>(x, f);
  //computeK2_F5<<< ps, 128 >>>(x, f);
  checkCudaErrors(hipGetLastError());
}
