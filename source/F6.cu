#include "hip/hip_runtime.h"
#include "F6.cuh"
#include "IO.h"
#include "constants.cuh"

#include <iostream>
#include <vector>
#include <iterator>
#include <fstream>

/*
 * Shifted and Rotated Griewank's Function
 *
 * as defined in "Problem Definitions and Evaluation Criteria for the
 * CEC 2013 Special Session and Competition on Real-Parameter Optimization",
 * by Liang, J.J., Qu, B.-Y., Suganthan, P.N., Hernandez-Diaz, A.G.,
 * Computational Intelligence Laboratory, Zhengzhou University, Zhengzhou,
 * China and Nanyang Technological University, Singapore, Technical Report,
 * v. 2012, p. 3-18, 2013.
*/

F6::F6(uint _dim, uint _ps):Benchmarks()
{
  n_dim = _dim;
  ps = _ps;
  min = -600.0;
  max = +600.0;
  ID = 6;

  n_threads = 32;
  n_blocks = (ps%n_threads)? (ps/n_threads)+1 : (ps/n_threads);

  /* ---------------------------------------------- */
  /* Load a shift vector to test the bench function */
  std::string file_name = "data-files/shift_griewank.mat";
  std::string vec_name = "Shift - Griewank [-600.0, +600.0]";
  IO * io = new IO();
  std::ifstream file(file_name);
  if( not file.is_open() ){
    std::cout << "\"data-files/shift_griewank.mat\" could not be opened\n";
    exit(-1);
  }
  auto loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(shift), (void *) loaded_vec.data(), n_dim * sizeof(float)));

  /* ---------------------------------------------- */
  /* Load a rotate matrix                           */
  if(!(n_dim==2 or n_dim==5 or n_dim==10 or n_dim==20 or n_dim==30 or n_dim==50 or n_dim==100)){
    printf("\nError: Rotation matrix are only defined for D = 2,5,10,20,30,50,100.\n");
    exit(-1);
  }

  file_name = "data-files/rot/M_D" + std::to_string(n_dim) + ".txt";
  vec_name = "M_D" + std::to_string(n_dim);
  file.open(file_name, std::ifstream::in);
  if( not file.is_open() ){
    std::cout << "Error opening rotation matrix file\n";
    exit(-1);
  }
  loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_rotation), (void *) loaded_vec.data(), n_dim * n_dim * sizeof(float)));

}

F6::~F6()
{
  /*empty*/
}

__global__ void computeK_F6(float * x, float * f){
  uint id_p = threadIdx.x + (blockIdx.x * blockDim.x);
  uint ps = params.ps;
  if( id_p < ps ){
    uint ndim = params.n_dim;
    uint id_d = id_p * ndim;
    uint i, j, k = ndim - 1;

    //The constant 600/100 is needed because on rotate operation
    //the value of a dimension can be higher than bounds;

    float z[100];
    //shift
    for( i = 0; i < ndim; i++ )
      z[i] = (x[id_d + i] - shift[i]) * 6.0;

    float z_rot[100];
    //rotation
    for( i = 0; i < ndim; i++ ){
      z_rot[i] = 0.0;
      for( j = 0; j < ndim; j++ )
        z_rot[i] += z[j] * m_rotation[i * ndim + j];
    }

    float s = 0.0, p = 1.0;
    for(uint i = 0; i < ndim; i++){
      z[i] = z_rot[i] * __powf(100.0, 1.0*i/k/2.0);

      s += z[i] * z[i];
      p *= __cosf( z[i] / __fsqrt_rn(1.0+i) );
    }
    s = 1.0 + s/4000.0 - p;

    if( s <= 10e-08 )
      s = 0.0;

    f[id_p] = s;
  }
}

void F6::compute(float * x, float * f){
  computeK_F6<<< n_blocks, n_threads >>>(x, f);
  checkCudaErrors(hipGetLastError());
}
