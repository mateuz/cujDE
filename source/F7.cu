#include "hip/hip_runtime.h"
#include "F7.cuh"
#include "IO.h"
#include "constants.cuh"

#include <iostream>
#include <vector>
#include <iterator>
#include <fstream>

/*
 * Shifted and Rotated Rastrigin's Function
 *
 * as defined in "Problem Definitions and Evaluation Criteria for the
 * CEC 2013 Special Session and Competition on Real-Parameter Optimization",
 * by Liang, J.J., Qu, B.-Y., Suganthan, P.N., Hernandez-Diaz, A.G.,
 * Computational Intelligence Laboratory, Zhengzhou University, Zhengzhou,
 * China and Nanyang Technological University, Singapore, Technical Report,
 * v. 2012, p. 3-18, 2013.
*/

F7::F7(uint _dim, uint _ps):Benchmarks()
{
  n_dim = _dim;
  ps = _ps;
  min = -100.0;
  max = +100.0;
  ID = 7;

  n_threads = 32;
  n_blocks = (ps%n_threads)? (ps/n_threads)+1 : (ps/n_threads);

  /* ---------------------------------------------- */
  /* Load a shift vector to test the bench function */
  std::string file_name = "data-files/shift_rastrigin.mat";
  std::string vec_name = "Shift - Griewank";
  IO * io = new IO();
  std::ifstream file(file_name);
  if( not file.is_open() ){
    std::cout << "\"data-files/shift_rastrigin.mat\" could not be opened\n";
    exit(-1);
  }
  auto loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(shift), (void *) loaded_vec.data(), n_dim * sizeof(float)));

  /* ---------------------------------------------- */
  /* Load a rotate matrix                           */
  if(!(n_dim==2 or n_dim==5 or n_dim==10 or n_dim==20 or n_dim==30 or n_dim==50 or n_dim==100)){
    printf("\nError: Rotation matrix are only defined for D = 2,5,10,20,30,50,100.\n");
    exit(-1);
  }

  file_name = "data-files/rot/M_D" + std::to_string(n_dim) + ".txt";
  vec_name = "M_D" + std::to_string(n_dim);
  file.open(file_name, std::ifstream::in);
  if( not file.is_open() ){
    std::cout << "Error opening rotation matrix file\n";
    exit(-1);
  }
  loaded_vec = io->load_vector<float>( vec_name, file ) ;
  file.close();
  /* ---------------------------------------------- */

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(m_rotation), (void *) loaded_vec.data(), n_dim * n_dim * sizeof(float)));

}

F7::~F7()
{
  /*empty*/
}

__global__ void computeK_F7(float * x, float * f){
  /* future implementation */
}

void F7::compute(float * x, float * f){
  computeK_F7<<< n_blocks, n_threads >>>(x, f);
  checkCudaErrors(hipGetLastError());
}
